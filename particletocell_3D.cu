#include "hip/hip_runtime.h"
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

// For each grid cell, reduce over subcells
__global__ void reduction_cells(double *w, int r, int RM_total, int M_total)
{
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    int j;
    int tid = threadIdx.x;

    // If valid grid cell index
    if(i < M_total) {

        // create memory to store temp sums and load elements
        extern __shared__ double temp_sum[];

        // move the w values into tempsum for this thread
        for (j = 0; j < r; j++) {
            temp_sum[r*tid + j] = w[r*i + j];
        }
        __syncthreads();
   
        // for each grid cell, do the sum, putting the value ultimately into the "0" idx
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {

            if (tid < s) {
                for(j = 0; j < r; j++) {
                    temp_sum[r * tid + j] += temp_sum[r * (tid + s) + j];
                }
            }

            // the values for r for this cell are now fully summed
            __syncthreads();
        }

        // each block writes uses thread 0 to write
        // results back into w from temp_sum
        if (tid == 0){

            for(j = 0; j < r; j++) {
                w[r*blockIdx.x+j] = temp_sum[j];
            }

        }

    }
}


// Does the atomic reductions for each subcell
__global__ void reduction_subcells(double *w, double *v, double *x0, double *x1,
                                   double *x2, int r, int Nc, int M0, int M1,
                                   int M2, int M_total) 
{

    int i=threadIdx.x + (blockIdx.x * blockDim.x);
    int j=threadIdx.y + (blockIdx.y * blockDim.y);

    // if valid particle and valid data point indexes
    if(i < Nc && j < r) {

        // Grab 3D bin of particle i mapped to 1D
        int b = M1*M2*int(x0[i]*float(M0)) + M2*int(x1[i]*float(M1)) + int(x2[i]*float(M2));

        // Atomically add in the value j for particle i
        atomicAdd(&w[r*b+j] , v[r*i+j]);

    }

}


// Main function called by cuda_red3D

/*
Performs 3D particle-to-cell summation for a set of 
Nc particles with weights dw_ar and 3D positions dxD_ar
and r different summation values dv_ar

-- Nc: particle count 
-- r: num data points per particle
-- MD: # grid cells in dimension D
-- RMD: # total subcells in dimension D
-- dxD_ar: dimension D normalized positions for particles (len Nc each)
-- dw_ar: 1D weight arrary for particles (len Nc)
-- dv_ar: 1D array of particle data points (len r*Nc)

*/
void calls(int Nc, int r,
           int M0, int M1, int M2,
           int RM0, int RM1, int RM2,
           py::array_t<double>& dw_ar, 
           py::array_t<double>& dv_ar, 
           py::array_t<double>& dx0_ar, 
           py::array_t<double>& dx1_ar, 
           py::array_t<double>& dx2_ar)
{

    // Total grid sizes
    int M_total = M0*M1*M2;
    int RM_total = RM0*RM1*RM2;

    // Setup for 1st kernel //
    int blockx = 256; // split up particles into chunks of 256
    int blocky = r; // to match num. values
    dim3 tpb(blockx, blocky, 1);
    dim3 bpg(ceil(Nc/blockx) + 1, ceil(r / blocky) + 1, 1);

    // Setup for 2nd kernel // 
    // num. values per cell = num. values * subells per cell
    int bloks = ceil((double) M_total/RM_total ); 
    size_t ns = bloks * r * sizeof(double); // memory for temp arrays
    dim3 tpb2(bloks, 1, 1);
    dim3 bpg2(ceil(M_total / bloks) + 1, 1, 1);

    // Recasting the arrays from Python
    double* dw  = static_cast<double *>(dw_ar.request().ptr);
    double* dv  = static_cast<double *>(dv_ar.request().ptr);
    double* dx0 = static_cast<double *>(dx0_ar.request().ptr);
    double* dx1 = static_cast<double *>(dx1_ar.request().ptr);
    double* dx2 = static_cast<double *>(dx2_ar.request().ptr);

    // Call the subcell reductions
    reduction_subcells<<<bpg, tpb>>>(dw, dv, dx0, dx1, dx2, r, Nc, M0, M1, M2, M_total);

    // If multiple subcells used per cell, do local summations
    if ((M_total / RM_total) > 1){
        reduction_cells<<<bpg2, tpb2, ns>>>(dw, r, RM_total, M_total);
    }

}

PYBIND11_MODULE(cuda_red3D, m) {
    m.def("calls", &calls);
}



